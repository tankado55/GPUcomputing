#include "hip/hip_runtime.h"

#include <assert.h>
#include "../utils/common.h"



/*
 *  Block by block parallel implementation with warp unrolling
 */
__global__ void blockParReduceUroll(int *in, int *out, ulong n) {

	
	//TODO
	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int *thisBlock = in + blockIdx.x * blockDim.x;

	// in-place reduction in global memory
	for (uint stride = blockDim.x / 2; stride > 32; stride >>= 1)  {
		if (tid < stride)
			thisBlock[tid] += thisBlock[tid + stride];

		// synchronize within threadblock
		__syncthreads();
	}

	// warp unrolling
	if (tid < 32) {
		volatile int *vmem = thisBlock;
		vmem[tid] += vmem[tid + 32];
		vmem[tid] += vmem[tid + 16];
		vmem[tid] += vmem[tid + 8];
		vmem[tid] += vmem[tid + 4];
		vmem[tid] += vmem[tid + 2];
		vmem[tid] += vmem[tid + 1];
	}

	// write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}

/*
 *  Multi block parallel implementation with block and warp unrolling
 */
__global__ void multBlockParReduceUroll8(int *in, int *out, ulong n) {

	
	//TODO
    uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int *thisBlock = in + blockIdx.x * blockDim.x * 8;

	// in-place reduction in global memory
	
	for (int i = 0; i < 7; ++i) {		
		thisBlock[tid] += thisBlock[tid + blockDim.x + (blockDim.x * i)];				
	}

	__syncthreads();

	for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)  {		
		if (tid < stride) {
			if (idx + stride < n) {
				thisBlock[tid] += thisBlock[tid + stride];
			}
			
		}

		// synchronize within threadblock
		__syncthreads();
	}

	// warp unrolling
	if (tid < 32) {
		volatile int *vmem = thisBlock;
		vmem[tid] += vmem[tid + 32];
		vmem[tid] += vmem[tid + 16];
		vmem[tid] += vmem[tid + 8];
		vmem[tid] += vmem[tid + 4];
		vmem[tid] += vmem[tid + 2];
		vmem[tid] += vmem[tid + 1];
	}
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}

/*
 *  Multi block parallel implementation with block and warp unrolling
 */
__global__ void multBlockParReduceUroll16(int *in, int *out, ulong n) {

	
	// TODO
	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x * 16 + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int *thisBlock = in + blockIdx.x * blockDim.x * 16;

	// in-place reduction in global memory
	
	for (int i = 0; i < 15; ++i) {		
		thisBlock[tid] += thisBlock[tid + blockDim.x + (blockDim.x * i)];				
	}

	__syncthreads();

	for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)  {		
		if (tid < stride) {
			if (idx + stride < n) {
				thisBlock[tid] += thisBlock[tid + stride];
			}
			
		}

		// synchronize within threadblock
		__syncthreads();
	}

	// warp unrolling
	if (tid < 32) {
		volatile int *vmem = thisBlock;
		vmem[tid] += vmem[tid + 32];
		vmem[tid] += vmem[tid + 16];
		vmem[tid] += vmem[tid + 8];
		vmem[tid] += vmem[tid + 4];
		vmem[tid] += vmem[tid + 2];
		vmem[tid] += vmem[tid + 1];
	}
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void blockParReduce2(int *in, int *out, ulong n) {

	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int *thisBlock = in + blockIdx.x * blockDim.x;

	// in-place reduction in global memory
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)  {
		if (tid < stride)
			thisBlock[tid] += thisBlock[tid + stride];

		// synchronize within threadblock
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}



/*
 * MAIN: test on parallel reduction
 */
int main(void) {
	int *a, *b, *d_a, *d_b;
	int blockSize = 1024;            // block dim 1D
	ulong numBlock = 1024*1024;      // grid dim 1D
	ulong n = blockSize * numBlock;  // array dim
	long sum_CPU = 0, sum_GPU;
	long nByte = n*sizeof(int), mByte = numBlock * sizeof(int);
	double start, stopGPU, stopCPU, speedup;

	printf("\n****  test on parallel reduction  ****\n");

	// init
	a = (int *) malloc(nByte);
	b = (int *) malloc(mByte);
	CHECK(hipMalloc((void **) &d_a, nByte));
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void **) &d_b, mByte));
	CHECK(hipMemset((void *) d_b, 0, mByte));

	/***********************************************************/
	/*                     CPU reduction                       */
	/***********************************************************/
	printf("  Vector length: %.2f GB\n",n/(1024.0*1024.0*1024.0));
	printf("\n  CPU procedure...\n");
	start = seconds();
	for (ulong i = 0; i < n; i++) sum_CPU += a[i];
	stopCPU = seconds() - start;
	printf("    Elapsed time: %f (sec) \n", stopCPU);
	printf("    sum: %lu\n",sum_CPU);

	printf("\n  GPU kernels (mem required %lu bytes)\n", nByte);

	/***********************************************************/
	/*        KERNEL blockParReduce2  (non divergent)          */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: blockParReduce2...\n");
	start = seconds();
	blockParReduce2<<<numBlock, blockSize>>>(d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU/stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU,speedup);
	CHECK(hipGetLastError());
  // memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
  // check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++) {
		sum_GPU += b[i];
	}
	assert(sum_GPU == n);
  // reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*               KERNEL blockParReduceUroll                */
	/***********************************************************/
	
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: blockParReduceUroll...\n");
	start = seconds();
	blockParReduceUroll<<<numBlock, blockSize>>>(d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU/stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU,speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);
	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*            KERNEL multBlockParReduceUroll8              */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: multBlockParReduceUroll8...\n");
	start = seconds();
	multBlockParReduceUroll8<<<numBlock/8, blockSize>>>(d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU/stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU,speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock/8; i++)
		sum_GPU += b[i];
	printf("    sum: %lu\n",sum_GPU);
	printf("    0: %d 1: %d\n", b[0], b[1]);
	assert(sum_GPU == n);
	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*            KERNEL multBlockParReduceUroll16             */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: multBlockParReduceUroll16...\n");
	start = seconds();
	multBlockParReduceUroll16<<<numBlock/16, blockSize>>>(d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU/stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU,speedup);
	CHECK(hipGetLastError());
	// memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock/16; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);

	hipFree(d_a);

	CHECK(hipDeviceReset());
	return 0;
}
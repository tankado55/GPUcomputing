#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "/content/GPUcomputing/utils/common.h"


/*
 *  Block by block parallel implementation with divergence (sequential schema)
 */
__global__ void blockParReduce1(int *in, int *out, ulong n) {

	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int *thisBlock = in + blockIdx.x * blockDim.x;

	// in-place reduction in global memory
	for (int stride = 1; stride < blockDim.x; stride *= 2) {
		if ((tid % (2 * stride)) == 0)
			thisBlock[tid] += thisBlock[tid + stride];

		// synchronize within threadblock
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}

/*
 *  Block by block parallel implementation without divergence (interleaved schema)
 */
__global__ void blockParReduce2(int *in, int *out, ulong n) {

	uint tid = threadIdx.x;
	ulong idx = blockIdx.x * blockDim.x + threadIdx.x;

	// boundary check
	if (idx >= n)
		return;

	// convert global data pointer to the local pointer of this block
	int *thisBlock = in + blockIdx.x * blockDim.x;

	// in-place reduction in global memory
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)  {
		if (tid < stride)
			thisBlock[tid] += thisBlock[tid + stride];

		// synchronize within threadblock
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0)
		out[blockIdx.x] = thisBlock[0];
}


/*
 * MAIN: test on parallel reduction
 */
int main(void) {
	int *a, *b, *d_a, *d_b;
	int blockSize = 1024;            // block dim 1D
	ulong numBlock = 1024*1024;      // grid dim 1D
	ulong n = blockSize * numBlock;  // array dim
	long sum_CPU = 0, sum_GPU;
	long nByte = n*sizeof(int), mByte = numBlock * sizeof(int);
	double start, stopGPU, stopCPU, speedup;

	printf("\n****  test on parallel reduction  ****\n");

	// init
	a = (int *) malloc(nByte);
	b = (int *) malloc(mByte);
	for (ulong i = 0; i < n; i++) a[i] = 1;

	CHECK(hipMalloc((void **) &d_a, nByte));
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));
	CHECK(hipMalloc((void **) &d_b, mByte));
	CHECK(hipMemset((void *) d_b, 0, mByte));

	/***********************************************************/
	/*                     CPU reduction                       */
	/***********************************************************/
	printf("  Vector length: %.2f MB\n",n/(1024.0*1024.0));
	printf("\n  CPU procedure...\n");
	start = seconds();
	for (ulong i = 0; i < n; i++) 
    sum_CPU += a[i];
	stopCPU = seconds() - start;
	printf("    Elapsed time: %f (sec) \n", stopCPU);
	printf("    sum: %lu\n",sum_CPU);

	printf("\n  GPU kernels (mem required %lu bytes)\n", nByte);

	/***********************************************************/
	/*         KERNEL blockParReduce1 (divergent)              */
	/***********************************************************/
	// block by block parallel implementation with divergence
	printf("\n  Launch kernel: blockParReduce1...\n");
	start = seconds();
	blockParReduce1<<<numBlock, blockSize>>>(d_a, d_b, n);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU/stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU,speedup);
	
  // memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	
  // check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);

	// reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i]=1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	/***********************************************************/
	/*        KERNEL blockParReduce2  (non divergent)          */
	/***********************************************************/
	// block by block parallel implementation without divergence
	printf("\n  Launch kernel: blockParReduce2...\n");
	start = seconds();
	blockParReduce2<<<numBlock, blockSize>>>(d_a, d_b, n);
	CHECK(hipDeviceSynchronize());
	stopGPU = seconds() - start;
	speedup = stopCPU/stopGPU;
	printf("    Elapsed time: %f (sec) - speedup %.1f\n", stopGPU,speedup);
	CHECK(hipGetLastError());
	
  // memcopy D2H
	CHECK(hipMemcpy(b, d_b, mByte, hipMemcpyDeviceToHost));
	
  // check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++) {
		sum_GPU += b[i];
  //		printf("b[%d] = %d\n",i,b[i]);
	}
	assert(sum_GPU == n);
	
  // reset input vector on GPU
	for (ulong i = 0; i < n; i++) a[i] = 1;
	CHECK(hipMemcpy(d_a, a, nByte, hipMemcpyHostToDevice));

	// check result
	sum_GPU = 0;
	for (uint i = 0; i < numBlock; i++)
		sum_GPU += b[i];
	assert(sum_GPU == n);

	hipFree(d_a);

	CHECK(hipDeviceReset());
	return 0;
}

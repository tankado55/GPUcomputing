#include "hip/hip_runtime.h"

#include <stdio.h>
#include "../utils/common.h"

#define PI 3.141592f
#define NSTREAM 4

/*
 * Kernel: tabular function
 */
__global__ void tabular(float *a, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < n) {
		float x = PI * (float)i / (float)n;
		float s = sinf(x);
		float c = cosf(x);
		a[i] = sqrtf(abs(s * s - c * c));
	}
}

/*
 * Kernel: tabular function using streams
 */
__global__ void tabular_streams(float *a, int n, int offset) {
	int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  if (i < n) {
    float x = PI * (float)i / (float)n;
    float s = sinf(x);
    float c = cosf(x);
    a[i] = sqrtf(abs(s * s - c * c));
  }
}

/*
 * Error measure
 */
float maxError(float *a, int n) {
	float maxE = 0;
	for (int i = 0; i < n; i++) {
		float error = fabs(a[i] - 1.0f);
		if (error > maxE)
			maxE = error;
	}
	return maxE;
}

/*
 * Main: tabular function
 */
int main(void) {

    dim3 grid  ((nElem + block.x - 1) / block.x);
	
  // main params
    uint MB = 1024*1024;
    uint nElem = 256*MB;
    int blockSize = 256;
    size_t nBytes = nElem * sizeof(float);
    int iElem = nElem / NSTREAM;
    size_t iBytes = iElem * sizeof(float);

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

	// streams
    hipStream_t stream[NSTREAM];
	
	// allocate pinned host memory and device memory
    float *h_A, *hostRef, *gpuRef;
    CHECK(hipHostAlloc((void**)&h_A, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&gpuRef, nBytes, hipHostMallocDefault));
    CHECK(hipHostAlloc((void**)&hostRef, nBytes, hipHostMallocDefault));

    float *d_A;
    CHECK(hipMalloc((float**)&d_A, nBytes));
	
	// create events and streams
    for (int i = 0; i < NSTREAM; ++i)
        CHECK(hipStreamCreate(&stream[i]));

    CHECK(hipEventRecord(start, 0));
	
	// baseline case - sequential transfer and execute
    
	// asynchronous version 1: loop over {copy, kernel, copy}
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&d_A[ioffset], &h_A[ioffset], iBytes, hipMemcpyHostToDevice, stream[i]));
    }
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        tabular_streams<<<grid, block, 0, stream[i]>>>(&d_A[ioffset], iElem, ioffset);
    }

    // enqueue asynchronous transfers from the device
    for (int i = 0; i < NSTREAM; ++i) {
        int ioffset = i * iElem;
        CHECK(hipMemcpyAsync(&gpuRef[ioffset], &d_A[ioffset], iBytes, hipMemcpyDeviceToHost, stream[i]));
    }

    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));
    float execution_time;
    CHECK(hipEventElapsedTime(&execution_time, start, stop));

    printf("\n");
    printf("Actual results from overlapped data transfers:\n");
    printf(" overlap with %d streams : %f ms (%f GB/s)\n", NSTREAM, execution_time, (nBytes * 2e-6) / execution_time );
    printf(" speedup                : %f \n", ((itotal - execution_time) * 100.0f) / itotal);
	
	// asynchronous version 2: loop over copy, loop over kernel, loop over copy
	
	// cleanup

    // free device global memory
  CHECK(hipFree(d_A));

  // free host memory
  CHECK(hipHostFree(h_A));
  CHECK(hipHostFree(hostRef));
  CHECK(hipHostFree(gpuRef));

  // destroy events
  CHECK(hipEventDestroy(start));
  CHECK(hipEventDestroy(stop));

  // destroy streams
  for (int i = 0; i < NSTREAM; ++i)
    CHECK(hipStreamDestroy(stream[i]));

  CHECK(hipDeviceReset());
	
	return 0;
}

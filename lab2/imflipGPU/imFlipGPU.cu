#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bmpUtil.h"
#include "../../utils/common.h"

/*
 * Kernel 1D that flips the given image vertically
 * each thread only flips a single pixel (R,G,B)
 */
__global__ void VflipGPU(pel *imgDst, const pel *imgSrc, const uint w, const uint h) {
	// ** pixel granularity **
	uint i = blockIdx.x;               // block ID
	uint j = threadIdx.x;              // thread ID
	uint b = blockDim.x;               // block dim
	uint x = b * i + j;                // 1D pixel linear index
	uint m = (w + b - 1) / b;          // num of blocks in a row
	uint r = i / m;                    // row of the source pixel
	uint c = x - r * w;                // col of the source pixel

	if (c >= w)                        // col out of range
		return;


	//  ** byte granularity **
	uint s = (w * 3 + 3) & (~3);       // num bytes x row (mult. 4)
	uint r1 = h - 1 - r;               // dest. row (mirror)
	uint p = s * r + 3*c;              // src byte position of the pixel
	uint q = s * r1 + 3*c;             // dst byte position of the pixel
	// swap pixels RGB
	imgDst[q] = imgSrc[p];             // R
	imgDst[q + 1] = imgSrc[p + 1];     // G
	imgDst[q + 2] = imgSrc[p + 2];     // B
}

/*
 *  Kernel that flips the given image horizontally
 *  each thread only flips a single pixel (R,G,B)
 */
__global__ void HflipGPU(pel *ImgDst, pel *ImgSrc, uint width) {
	uint b = blockDim.x;
	uint i = blockIdx.x;
	uint j = threadIdx.x;
	uint x = b * i + j;
	uint w = (width + b - 1) / b;  // ceil
	uint s = (width * 3 + 3) & (~3);
	uint r = i / w;
	uint c = x - r * w * b;

	if (c >= width)
		return;			// col out of range

	uint c1 = width - 1 - c;
	uint p = r * s;
	uint sr = p + 3 * c;
	uint ds = p + 3 * c1;

	// swap pixels RGB   @c , @c1
	ImgDst[ds] = ImgSrc[sr];
	ImgDst[ds + 1] = ImgSrc[sr + 1];
	ImgDst[ds + 2] = ImgSrc[sr + 2];
}

/*
 *  Read a 24-bit/pixel BMP file into a 1D linear array.
 *  Allocate memory to store the 1D image and return its pointer
 */
pel *ReadBMPlin(char* fn) {
	static pel *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL) {
		printf("\n\n%s NOT FOUND\n\n", fn);
		exit(EXIT_FAILURE);
	}

	pel HeaderInfo[54];
	size_t nByte = fread(HeaderInfo, sizeof(pel), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*) &HeaderInfo[18];
	img.width = width;
	int height = *(int*) &HeaderInfo[22];
	img.height = height;
	int s = (width * 3 + 3) & (~3);  // row is multiple of 4 pixel
	img.rowByte = s;
	//save header for re-use
	memcpy(img.headInfo, HeaderInfo, 54);
	printf("\n Input File name: %5s  (%d x %d)   File Size=%lu", fn, img.width,
			img.height, IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img = (pel *) malloc(IMAGESIZE);
	if (Img == NULL)
		return Img;      // Cannot allocate memory
	// read the image from disk
	size_t out = fread(Img, sizeof(pel), IMAGESIZE, f);
	fclose(f);
	return Img;
}

/*
 *  Write the 1D linear-memory stored image into file
 */
void WriteBMPlin(pel *Img, char* fn) {
	FILE* f = fopen(fn, "wb");
	if (f == NULL) {
		printf("\n\nFILE CREATION ERROR: %s\n\n", fn);
		exit(1);
	}
	//write header
	fwrite(img.headInfo, sizeof(pel), 54, f);
	//write data
	fwrite(Img, sizeof(pel), IMAGESIZE, f);
	printf("\nOutput File name: %5s  (%u x %u)   File Size=%lu", fn, img.width,
			img.height, IMAGESIZE);
	fclose(f);
}

/*
 * MAIN
 */
int main(int argc, char **argv) {
	char flip = 'V';
	uint b = 256, dimGrid;
	pel *imgSrc, *imgDst;		 // Where images are stored in CPU
	pel *imgSrcGPU, *imgDstGPU;	 // Where images are stored in GPU

	if (argc > 4) {
		b = atoi(argv[4]);
		flip = argv[3][0];
	}
	else if (argc > 3) {
		flip = argv[3][0];
	}
	else if (argc < 3) {
		printf("\n\nUsage:   imflipGPU InputFilename OutputFilename [V/H] [b]\n");
		exit(EXIT_FAILURE);
	}
	if ((flip != 'V') && (flip != 'H')) {
		printf("Invalid flip option '%c'. Must be 'V','H'... \n",flip);
		exit(EXIT_FAILURE);
	}

	// Create CPU memory to store the input and output images
	imgSrc = ReadBMPlin(argv[1]); // Read the input image if memory can be allocated
	if (imgSrc == NULL) {
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	imgDst = (pel *) malloc(IMAGESIZE);
	if (imgDst == NULL) {
		free(imgSrc);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	// Allocate GPU buffer for the input and output images
	CHECK(hipMalloc((void**) &imgSrcGPU, IMAGESIZE));
	CHECK(hipMalloc((void**) &imgDstGPU, IMAGESIZE));

	// Copy input vectors from host memory to GPU buffers.
	CHECK(hipMemcpy(imgSrcGPU, imgSrc, IMAGESIZE, hipMemcpyHostToDevice));

	// invoke kernels (define grid and block sizes)
	int rowBlock = (WIDTH + b - 1) / b;
	dimGrid = HEIGHT * rowBlock;


	double start = seconds();   // start time

	switch (flip) {
	case 'H':
		HflipGPU<<<dimGrid, b>>>(imgDstGPU, imgSrcGPU, WIDTH);
		break;
	case 'V':
		VflipGPU<<<dimGrid, b>>>(imgDstGPU, imgSrcGPU, WIDTH, HEIGHT);
		break;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CHECK(hipDeviceSynchronize());

	double stop = seconds();   // elapsed time

	// Copy output (results) from GPU buffer to host (CPU) memory.
	CHECK(hipMemcpy(imgDst, imgDstGPU, IMAGESIZE, hipMemcpyDeviceToHost));

	// Write the flipped image back to disk
	WriteBMPlin(imgDst, argv[2]);

	printf("\nKernel elapsed time %f sec \n\n", stop - start);

	// Deallocate CPU, GPU memory and destroy events.
	hipFree(imgSrcGPU);
	hipFree(imgDstGPU);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools spel as Parallel Nsight and Visual Profiler to show complete traces.
	hipError_t	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(imgSrc);
		free(imgDst);
		exit(EXIT_FAILURE);
	}
	free(imgSrc);
	free(imgDst);
	return (EXIT_SUCCESS);
}







    



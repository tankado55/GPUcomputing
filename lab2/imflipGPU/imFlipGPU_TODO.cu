#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bmpUtil.h"
#include "../../utils/common.h"

/*
 * Kernel 1D that flips the given image vertically
 * each thread only flips a single pixel (R,G,B)
 */
__global__ void VflipGPU(pel *imgDst, const pel *imgSrc, const uint w, const uint h) {
	// TODO
}

/*
 *  Kernel that flips the given image horizontally
 *  each thread only flips a single pixel (R,G,B)
 */
__global__ void HflipGPU(pel *ImgDst, pel *ImgSrc, uint width) {
	//TODO
}

/*
 *  Read a 24-bit/pixel BMP file into a 1D linear array.
 *  Allocate memory to store the 1D image and return its pointer
 */
pel *ReadBMPlin(char* fn) {
	static pel *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL) {
		printf("\n\n%s NOT FOUND\n\n", fn);
		exit(EXIT_FAILURE);
	}

	pel HeaderInfo[54];
	size_t nByte = fread(HeaderInfo, sizeof(pel), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*) &HeaderInfo[18];
	img.width = width;
	int height = *(int*) &HeaderInfo[22];
	img.height = height;
	int s = (width * 3 + 3) & (~3);  // row is multiple of 4 pixel
	img.rowByte = s;
	//save header for re-use
	memcpy(img.headInfo, HeaderInfo, 54);
	printf("\n Input File name: %5s  (%d x %d)   File Size=%lu", fn, img.width,
			img.height, IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img = (pel *) malloc(IMAGESIZE);
	if (Img == NULL)
		return Img;      // Cannot allocate memory
	// read the image from disk
	size_t out = fread(Img, sizeof(pel), IMAGESIZE, f);
	fclose(f);
	return Img;
}

/*
 *  Write the 1D linear-memory stored image into file
 */
void WriteBMPlin(pel *Img, char* fn) {
	FILE* f = fopen(fn, "wb");
	if (f == NULL) {
		printf("\n\nFILE CREATION ERROR: %s\n\n", fn);
		exit(1);
	}
	//write header
	fwrite(img.headInfo, sizeof(pel), 54, f);
	//write data
	fwrite(Img, sizeof(pel), IMAGESIZE, f);
	printf("\nOutput File name: %5s  (%u x %u)   File Size=%lu", fn, img.width,
			img.height, IMAGESIZE);
	fclose(f);
}

/*
 * MAIN
 */
int main(int argc, char **argv) {
	char flip = 'V';
	uint b = 256, dimGrid;
	pel *imgSrc, *imgDst;		 // Where images are stored in CPU
	pel *imgSrcGPU, *imgDstGPU;	 // Where images are stored in GPU

	if (argc > 4) {
		b = atoi(argv[4]);
		flip = argv[3][0];
	}
	else if (argc > 3) {
		flip = argv[3][0];
	}
	else if (argc < 3) {
		printf("\n\nUsage:   imflipGPU InputFilename OutputFilename [V/H] [b]\n");
		exit(EXIT_FAILURE);
	}
	if ((flip != 'V') && (flip != 'H')) {
		printf("Invalid flip option '%c'. Must be 'V','H'... \n",flip);
		exit(EXIT_FAILURE);
	}

	// Create CPU memory to store the input and output images
	imgSrc = ReadBMPlin(argv[1]); // Read the input image if memory can be allocated
	if (imgSrc == NULL) {
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	imgDst = (pel *) malloc(IMAGESIZE);
	if (imgDst == NULL) {
		free(imgSrc);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	// Allocate GPU buffer for the input and output images
	CHECK(hipMalloc((void**) &imgSrcGPU, IMAGESIZE));
	CHECK(hipMalloc((void**) &imgDstGPU, IMAGESIZE));

	// Copy input vectors from host memory to GPU buffers.
	CHECK(hipMemcpy(imgSrcGPU, imgSrc, IMAGESIZE, hipMemcpyHostToDevice));

	// invoke kernels (define grid and block sizes)
	int rowBlock = (WIDTH + b - 1) / b;
	dimGrid = HEIGHT * rowBlock;


	double start = seconds();   // start time

	switch (flip) {
	case 'H':
		HflipGPU<<<dimGrid, b>>>(imgDstGPU, imgSrcGPU, WIDTH);
		break;
	case 'V':
		VflipGPU<<<dimGrid, b>>>(imgDstGPU, imgSrcGPU, WIDTH, HEIGHT);
		break;
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CHECK(hipDeviceSynchronize());

	double stop = seconds();   // elapsed time

	// Copy output (results) from GPU buffer to host (CPU) memory.
	CHECK(hipMemcpy(imgDst, imgDstGPU, IMAGESIZE, hipMemcpyDeviceToHost));

	// Write the flipped image back to disk
	WriteBMPlin(imgDst, argv[2]);

	printf("\nKernel elapsed time %f sec \n\n", stop - start);

	// Deallocate CPU, GPU memory and destroy events.
	hipFree(imgSrcGPU);
	hipFree(imgDstGPU);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools spel as Parallel Nsight and Visual Profiler to show complete traces.
	hipError_t	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(imgSrc);
		free(imgDst);
		exit(EXIT_FAILURE);
	}
	free(imgSrc);
	free(imgDst);
	return (EXIT_SUCCESS);
}







    



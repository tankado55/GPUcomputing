#include "hip/hip_runtime.h"
/**
 * hist.cu
 */
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <limits.h>

#include "../../utils/common.h"
#include "../../utils/BMP/ImageStuff.h"
#include "../../utils/BMP/bmpUtil.h"

/*
 * Kernel 1D that computes histogram on GPU
 */
__global__ void histogramBMP(uint *bins, const pel *imgSrc, const uint W, const uint N, const uint M) {
	
	// TODO
	uint id = blockIdx.x * blockDim.x + threadIdx.x;

	uint r = id / W; //row
	uint off = id - r * W; //col

	uint p = M * r + 3*off;
	pel R = imgSrc[p];
	pel G = imgSrc[p+1];
	pel B = imgSrc[p+2];
	atomicAdd(&bins[R], 1);
	atomicAdd(&bins[G+256], 1);
	atomicAdd(&bins[B+512], 1);
}

/*
 * Function that computes histogram on CPU
 */
void hist_CPU(uint *bins, const pel *imgSrc, const uint W, const uint H, const uint M) {
	for (int i = 0; i < W*H; i++) {
		uint r = i / W;              // row of the source pixel
		uint off = i - r * W;        // col of the source pixel

		//  ** byte granularity **
		uint p = M * r + 3*off;      // src byte position of the pixel
		pel R = imgSrc[p];
		pel G = imgSrc[p+1];
		pel B = imgSrc[p+2];
		bins[R] += 1;
		bins[G+256] += 1;
		bins[B+512] += 1;
	}
}

int main(int argc, char **argv) {

	uint dimBlock = 1024;
	pel *imgBMP_CPU;     // Where images are stored in CPU
	pel *imgBMP_GPU;	 // Where images are stored in GPU

	uint *binsRGB_CPU, *binsRGB_GPU, *binsRGB_GPU2CPU;
	uint N_bins = 3*256;
	uint bin_size = N_bins*sizeof(uint);

	if (argc > 2)
		dimBlock = atoi(argv[2]);
	else if (argc < 2) {
		printf("\n\nUsage:  hist InputFilename dimBlock\n");
		exit(EXIT_FAILURE);
	}

	// bins for CPU & GPU
	binsRGB_CPU = (uint*) calloc(N_bins, sizeof(uint));
	binsRGB_GPU2CPU = (uint*) malloc(bin_size);
	CHECK(hipMalloc((void**) &binsRGB_GPU, bin_size));

	// Create CPU memory to store the input image
	imgBMP_CPU = ReadBMPlin(argv[1]);
	if (imgBMP_CPU == NULL) {
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	// Allocate GPU buffer for image and bins
	CHECK(hipMalloc((void**) &imgBMP_GPU, IMAGESIZE));

	// Copy input vectors from host memory to GPU buffers.
	CHECK(hipMemcpy(imgBMP_GPU, imgBMP_CPU, IMAGESIZE, hipMemcpyHostToDevice));

	// CPU histogram
	double start = seconds();   // start time
	hist_CPU(binsRGB_CPU, imgBMP_CPU, WIDTH, HEIGHT, WIDTHB);
	double stop = seconds();   // elapsed time
	printf("\nCPU elapsed time %f sec \n\n", stop - start);

	// invoke kernels (define grid and block sizes)
	uint nPixels = WIDTH*HEIGHT;
	int dimGrid = (nPixels + dimBlock - 1) / dimBlock;
	printf("\ndimGrid = %d   dimBlock = %d\n",dimGrid,dimBlock);

	start = seconds();   // start time
	histogramBMP<<<dimGrid, dimBlock>>>(binsRGB_GPU, imgBMP_GPU, WIDTH, nPixels, WIDTHB);
	CHECK(hipDeviceSynchronize());
	stop = seconds();   // elapsed time
	printf("\nGPU elapsed time %f sec \n\n", stop - start);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	CHECK(hipMemcpy(binsRGB_GPU2CPU, binsRGB_GPU, bin_size, hipMemcpyDeviceToHost));

	for (int i = 0; i < N_bins/3; i++)
		printf("bin_GPU[%d] = \t%d\t%d\t%d\t -- bin_CPU[%d] = \t%d\t%d\t%d\n", i,
				binsRGB_GPU2CPU[i],binsRGB_GPU2CPU[i+256],binsRGB_GPU2CPU[i+512],
				i,binsRGB_CPU[i],binsRGB_CPU[i+256],binsRGB_CPU[i+512]);

	// Deallocate GPU memory
	hipFree(imgBMP_GPU);
	hipFree(binsRGB_GPU);

	// tracing tools spel as Parallel Nsight and Visual Profiler to show complete traces.
	CHECK(hipDeviceReset());

	return (EXIT_SUCCESS);
}

/*
 *  Read a 24-bit/pixel BMP file into a 1D linear array.
 *  Allocate memory to store the 1D image and return its pointer
 */
pel *ReadBMPlin(char* fn) {
	static pel *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL) {
		printf("\n\n%s NOT FOUND\n\n", fn);
		exit(EXIT_FAILURE);
	}

	pel HeaderInfo[54];
	size_t nByte = fread(HeaderInfo, sizeof(pel), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*) &HeaderInfo[18];
	img.width = width;
	int height = *(int*) &HeaderInfo[22];
	img.height = height;
	int RowBytes = (width * 3 + 3) & (~3);  // row is multiple of 4 pixel
	img.rowByte = RowBytes;
	//save header for re-use
	memcpy(img.headInfo, HeaderInfo, 54);
	printf("\n Input File name: %5s  (%d x %d)   File Size=%lu", fn, img.width, img.height, IMAGESIZE);

	// allocate memory to store the main image (1 Dimensional array)
	Img = (pel *) malloc(IMAGESIZE);
	if (Img == NULL)
		return Img;      // Cannot allocate memory
	// read the image from disk
	size_t out = fread(Img, sizeof(pel), IMAGESIZE, f);
	fclose(f);
	return Img;
}

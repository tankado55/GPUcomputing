#include "hip/hip_runtime.h"
<<<<<<< HEAD
#include <stdio.h>
#include <iostream>
#include <unistd.h>


using namespace std;

__global__ void helloFromGPU (void) {
  int tID = threadIdx.x;
  printf("Hello World from GPU (I'am thread %d)!\n", tID);
}

int main(void) {
<<<<<<< HEAD
  //# hello from GPU 
  cout << "Hello World from CPU!" << endl;
  hipSetDevice(0);
  helloFromGPU <<<1, 10>>>();
  hipDeviceSynchronize();
  sleep(10);
  return 0;
=======
    // hello from GPU 
    cout << "Hello World from CPU!" << endl;
    hipSetDevice(1);
    helloFromGPU <<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
>>>>>>> bd4ceea (a lot)
=======
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void helloFromGPU (void) {
    int tID = threadIdx.x;
    printf("Hello World from GPU (I'am thread = %d)!\n", tID);
}

int main(void) {
    // hello from GPU 
    cout << "Hello World from CPU!" << endl;
    hipSetDevice(0);
    helloFromGPU <<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
>>>>>>> main
}
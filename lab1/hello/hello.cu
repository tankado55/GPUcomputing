
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <unistd.h>


using namespace std;

__global__ void helloFromGPU (void) {
  int tID = threadIdx.x;
  printf("Hello World from GPU (I'am thread %d)!\n", tID);
}

int main(void) {
  //# hello from GPU 
  cout << "Hello World from CPU!" << endl;
  hipSetDevice(0);
  helloFromGPU <<<1, 10>>>();
  hipDeviceSynchronize();
  sleep(10);
  return 0;
}